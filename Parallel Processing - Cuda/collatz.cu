/*
Collatz code for CS 4380 / CS 5351

Copyright (c) 2019 Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is *not* permitted. Use in source and binary forms, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher


*/

#include <cstdio>
#include <algorithm>
#include <sys/time.h>
// 1 - Cuda Header File
#include <hip/hip_runtime.h>

// 2 - Thread count per block for the collatz code
static const int ThreadsPerBlock = 512;

// 3 - kernel function
static __global__ void collatz(int* maxlen, const long range)
{
  // compute sequence lengths
  // global identifier
  const long idx = threadIdx.x + blockIdx.x * (long)blockDim.x;

  //for (long i = 1; i <= range; i += 2) {
  if (idx%2 != 0) {
    long val = idx;
    int len = 1;
    while (val != 1) {
      len++;
      if ((val % 2) == 0) {
        val = val / 2;  // even
      } else {
        val = 3 * val + 1;  // odd
      }
    }

    //maxlen = std::max(maxlen, len);
  if ( *maxlen < len)
    atomicMax(maxlen, len);
  //}
  }
  //return maxlen;
}

static void CheckCuda()
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d: %s\n", e, hipGetErrorString(e));
    exit(-1);
  }
}

int main(int argc, char *argv[])
{
  printf("Collatz v1.1\n");

  // check command line
  if (argc != 2) {fprintf(stderr, "USAGE: %s range\n", argv[0]); exit(-1);}
  const long range = atol(argv[1]);
  if (range < 3) {fprintf(stderr, "ERROR: range must be at least 3\n"); exit(-1);}
  printf("range bound: %ld\n", range);

  // 10 - alloc space for device copy
  int* d_maxlen;
  const int size = sizeof(int);
  hipMalloc((void **)&d_maxlen, size);

  // 10 - alloc host copy
  int maxlen = 0;

  //  copy to device
  if (hipSuccess != hipMemcpy(d_maxlen, &maxlen, size, hipMemcpyHostToDevice)) {
     fprintf(stderr, "Error: failed to copy to device\n");
     exit(-1);
   }


  // start time
  timeval start, end;
  gettimeofday(&start, NULL);

  // call timed function
  // 11 - round up      8 - pass maxlen to kernel
  collatz<<<( range + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(d_maxlen, range);
  // 12 -
  hipDeviceSynchronize();

  // end time
  gettimeofday(&end, NULL);
  const double runtime = end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0;
  printf("compute time: %.4f s\n", runtime);
  // 13 - check fpr errors
  CheckCuda();

  // 14 - copy result back to host
  if (hipSuccess != hipMemcpy(&maxlen, d_maxlen, size, hipMemcpyDeviceToHost)) {
    fprintf(stderr, "Error: failed to copy from device\n");
    exit(-1);
  }
  printf("longest sequence: %d elements\n", maxlen);

  // 15 - free allocated memory
  hipFree(d_maxlen);

  return 0;
}
